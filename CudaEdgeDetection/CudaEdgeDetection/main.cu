#include "hip/hip_runtime.h"
// CudaEdgeDetection.cpp: define el punto de entrada de la aplicaci�n de consola.

#include "stdafx.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;
// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
 * Convolution funcion para cuda.  Destino tendra el mismo width/height como la fuente, 
 *
 * @param source      Source imagen host
 * @param width       Source imagen width
 * @param height      Source imagen height
 * @param paddingX    source imagen padding x 
 * @param paddingY    source imagen padding y
 * @param kOffset     offset en kernel almacenamiento de memoria constante
 * @param kWidth      kernel width
 * @param kHeight     kernel height
 * @param destination Imagen de destino
 */
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, unsigned int kOffset, int kWidth, int kHeight, unsigned char *destination)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    float sum = 0.0;
    int   pWidth = kWidth/2;
    int   pHeight = kHeight/2;

    //Solo ejecuta validos pixeles
    if(x >= pWidth+paddingX &&   y >= pHeight+paddingY &&  x < (blockDim.x * gridDim.x)-pWidth-paddingX &&
			 y < (blockDim.y * gridDim.y)-pHeight-paddingY)
    {
        for(int j = -pHeight; j <= pHeight; j++)
        {
            for(int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = (i+pWidth);
                int kj = (j+pHeight);
                float w  = convolutionKernelStore[(kj * kWidth) + ki + kOffset];

        
                sum += w * float(source[((y+j) * width) + (x+i)]);
            }
        }
    }
    
    // Promedio sum
    destination[(y * width) + x] = (unsigned char) sum;
}

__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    float af = float(a[idx]);
    float bf = float(b[idx]);

    c[idx] = (unsigned char) sqrtf(af*af + bf*bf);
}

// crea imagen buffer
unsigned char* createImageBuffer(unsigned int bytes, unsigned char **devicePtr)
{
    unsigned char *ptr = NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
    hipHostGetDevicePointer(devicePtr, ptr, 0);
    return ptr;
}


int main (int argc, char** argv){
    // Abre la camaraweb
    cv::VideoCapture camera(0);
    cv::Mat          frame;
    if(!camera.isOpened()) 
        return -1;
    
    // capture windows
    cv::namedWindow("Source");
    cv::namedWindow("Greyscale");
    cv::namedWindow("Blurred");
    cv::namedWindow("Sobel");

    // Funciones para obtener el tiempo de ejecucion 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Crea kernel gaussian(sum = 159)
    const float gaussianKernel5x5[25] = 
    {
        2.f/159.f,  4.f/159.f,  5.f/159.f,  4.f/159.f, 2.f/159.f,   
        4.f/159.f,  9.f/159.f, 12.f/159.f,  9.f/159.f, 4.f/159.f,   
        5.f/159.f, 12.f/159.f, 15.f/159.f, 12.f/159.f, 5.f/159.f,   
        4.f/159.f,  9.f/159.f, 12.f/159.f,  9.f/159.f, 4.f/159.f,   
        2.f/159.f,  4.f/159.f,  5.f/159.f,  4.f/159.f, 2.f/159.f,   
    };
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), gaussianKernel5x5, sizeof(gaussianKernel5x5), 0);
    const unsigned int gaussianKernel5x5Offset = 0;

    // Sobel gradient kernels
    const float sobelGradientX[9] =
    {
        -1.f, 0.f, 1.f,
        -2.f, 0.f, 2.f,
        -1.f, 0.f, 1.f,
    };
    const float sobelGradientY[9] =
    {
        1.f, 2.f, 1.f,
        0.f, 0.f, 0.f,
        -1.f, -2.f, -1.f,
    };
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientX, sizeof(sobelGradientX), sizeof(gaussianKernel5x5));
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientY, sizeof(sobelGradientY), sizeof(gaussianKernel5x5) + sizeof(sobelGradientX));
    const unsigned int sobelGradientXOffset = sizeof(gaussianKernel5x5)/sizeof(float);
    const unsigned int sobelGradientYOffset = sizeof(sobelGradientX)/sizeof(float) + sobelGradientXOffset;
 
    // Crea CPU/GPU imagenes compartidos
    camera >> frame;
    unsigned char *sourceDataDevice, *blurredDataDevice, *edgesDataDevice;
    cv::Mat source  (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &sourceDataDevice));
    cv::Mat blurred (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &blurredDataDevice));
    cv::Mat edges   (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &edgesDataDevice));

    // Crea 2 imagenes temporales (sobel gradients)
    unsigned char *deviceGradientX, *deviceGradientY;
    hipMalloc(&deviceGradientX, frame.size().width * frame.size().height);
    hipMalloc(&deviceGradientY, frame.size().width * frame.size().height);
    
    // Loop while captura imagenes
    while(1)
    {
        // Captura la imagen en escala de grises
        camera >> frame;
        cvtColor(frame, source, CV_BGR2GRAY);
        
        // Graba el tiempo que demora el proceso
        hipEventRecord(start);
        {
            // convolution kernel  parametros
            dim3 cblocks (frame.size().width / 16, frame.size().height / 16);
            dim3 cthreads(16, 16);

            // pythagoran kernel parametros
            dim3 pblocks (frame.size().width * frame.size().height / 256);
            dim3 pthreads(256, 1);

            //  gaussian blur (first kernel in store @ 0)
            convolve<<<cblocks,cthreads>>>(sourceDataDevice, frame.size().width, frame.size().height, 0, 0, gaussianKernel5x5Offset, 5, 5, blurredDataDevice);

            // sobel gradient convolutions (x&y padding is now 2 because there is a border of 2 around a 5x5 gaussian filtered image)
            convolve<<<cblocks,cthreads>>>(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientXOffset, 3, 3, deviceGradientX);
            convolve<<<cblocks,cthreads>>>(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientYOffset, 3, 3, deviceGradientY);
            pythagoras<<<pblocks,pthreads>>>(deviceGradientX, deviceGradientY, edgesDataDevice);

            hipDeviceSynchronize();
        }
        hipEventRecord(stop);
      
        // Muestra tiempo de ejecucion
        float ms = 0.0f;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        std::cout << "Elapsed GPU time: " << ms << " milliseconds" << std::endl;

        // Muestra resultados
        imshow("Source", frame);
        imshow("Greyscale", source);
        imshow("Blurred", blurred);
        imshow("Sobel", edges);

        // Spin
        if(cv::waitKey(1) == 27) break;
    }
    
    // Exit
    hipHostFree(source.data);
    hipHostFree(blurred.data);
    hipHostFree(edges.data);
    hipFree(deviceGradientX);
    hipFree(deviceGradientY);

    return 0;
}